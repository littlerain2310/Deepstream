#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstring>
#include <fstream>
#include <iostream>
#include <unordered_map>
#include "nvdsinfer_custom_impl.h"
#include "nvtx3/nvToolsExt.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_fp16.h>

static const int NUM_CLASSES_YOLO = 80;
#define OBJECTLISTSIZE 25200
#define BLOCKSIZE  1024
thrust::device_vector<NvDsInferParseObjectInfo> objects_v(OBJECTLISTSIZE);
thrust::device_vector<float> objects_floats(OBJECTLISTSIZE * (NUM_CLASSES_YOLO + 5));


extern "C" bool NvDsInferParseCustomYolo(
    std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList);


__global__ void decodeYoloTensor_cuda(NvDsInferParseObjectInfo *binfo/*output*/, void* data, int dimensions, int rows,
                                        int netW, int netH, float Threshold, bool is_fp16){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < rows) {
        float maxProb, bx, by, bw, bh, maxScore;
        int maxIndex;
        if (is_fp16){
            maxProb = __half2float(*(((__half*)data) + idx * dimensions + 4));
            //maxProb < Threshold, directly return
            if (maxProb < Threshold) {
                binfo[idx].detectionConfidence = 0;
                return;
            }
            bx = __half2float(*(((__half*)data) + idx * dimensions + 0));
            by = __half2float(*(((__half*)data) + idx * dimensions + 1));
            bw = __half2float(*(((__half*)data) + idx * dimensions + 2));
            bh = __half2float(*(((__half*)data) + idx * dimensions + 3));
            float class_score;
            maxScore = 0;
            maxIndex = 0;
            for (int j = 0; j < dimensions - 5; j++) {
                class_score = __half2float(*(((__half*)data) + idx * dimensions + 5 + j));
               if (class_score > maxScore) {
                  maxIndex = j;
                  maxScore = class_score;
               }
            }

        } else {
            maxProb = ((float*)data)[idx * dimensions + 4];
            //maxProb < Threshold, directly return
            if (maxProb < Threshold) {
                binfo[idx].detectionConfidence = 0;
                return;
            }
            bx = ((float*)data)[idx * dimensions + 0];
            by = ((float*)data)[idx * dimensions + 1];
            bw = ((float*)data)[idx * dimensions + 2];
            bh = ((float*)data)[idx * dimensions + 3];
            float * classes_scores = ((float*)data) + idx * dimensions + 5;
            maxScore = 0;
            maxIndex = 0;
            for (int j = 0; j < dimensions - 5; j++) {
               if (*classes_scores > maxScore) {
                  maxIndex = j;
                  maxScore = *classes_scores;
               }
               classes_scores++;
            }
        }
        float stride = 1.0;
        float xCenter = bx * stride;
        float yCenter = by * stride;
        float x0 = xCenter - bw / 2;
        float y0 = yCenter - bh / 2;
        float x1 = x0 + bw;
        float y1 = y0 + bh;
        x0 = fminf(netW, fmaxf(float(0.0), x0));
        y0 = fminf(netH, fmaxf(float(0.0), y0));
        x1 = fminf(netW, fmaxf(float(0.0), x1));
        y1 = fminf(netH, fmaxf(float(0.0), y1));
        binfo[idx].left = x0;
        binfo[idx].top = y0;
        binfo[idx].width = fminf(float(netW), fmaxf(float(0.0), x1-x0));
        binfo[idx].height = fminf(float(netH), fmaxf(float(0.0), y1-y0));
        binfo[idx].detectionConfidence = maxProb;
        binfo[idx].classId = maxIndex;
    }
    return;
}
static bool NvDsInferParseYolo(
    std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{

    if (outputLayersInfo.empty()) {
        std::cerr << "Could not find output layer in bbox parsing" << std::endl;;
        return false;
    }
    const NvDsInferLayerInfo &layer = outputLayersInfo[0];

    if (NUM_CLASSES_YOLO != detectionParams.numClassesConfigured)
    {
        std::cerr << "WARNING: Num classes mismatch. Configured:"
                  << detectionParams.numClassesConfigured
                  << ", detected by network: " << NUM_CLASSES_YOLO << std::endl;
    }
    float* data = (float*)layer.buffer;


    const int dimensions = layer.inferDims.d[1];
    int rows = layer.inferDims.numElements / layer.inferDims.d[1];

    int GRIDSIZE = ((OBJECTLISTSIZE-1)/BLOCKSIZE)+1;
    //find the min threshold
    float min_PreclusterThreshold = *(std::min_element(detectionParams.perClassPreclusterThreshold.begin(),
        detectionParams.perClassPreclusterThreshold.end()));
    decodeYoloTensor_cuda<<<GRIDSIZE,BLOCKSIZE>>>
        (thrust::raw_pointer_cast(objects_v.data()), data, dimensions, rows, networkInfo.width,
        networkInfo.height, min_PreclusterThreshold, layer.dataType == HALF);
    objectList.resize(OBJECTLISTSIZE);
    thrust::copy(objects_v.begin(),objects_v.end(),objectList.begin());//the same as cudamemcpy
    return true;
}

extern "C" bool NvDsInferParseCustomYolo(
    std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
    nvtxRangePush("NvDsInferParseYolo");
    bool ret = NvDsInferParseYolo (
        outputLayersInfo, networkInfo, detectionParams, objectList);

    nvtxRangePop();
    return ret;
}

/* Check that the custom function has been defined correctly */
CHECK_CUSTOM_PARSE_FUNC_PROTOTYPE(NvDsInferParseCustomYolo);